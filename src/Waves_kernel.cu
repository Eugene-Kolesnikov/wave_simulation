
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>

// complex math functions
__device__ float2 conjugate(float2 arg)
{
    return make_float2(arg.x, -arg.y);
}

__device__ float2 complex_exp(float arg)
{
    return make_float2(cosf(arg), sinf(arg));
}

__device__ float2 complex_add(float2 a, float2 b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 complex_mult(float2 ab, float2 cd)
{
    return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
}

// generate wave heightfield at time t based on initial heightfield and dispersion relationship
__global__ void generateSpectrumKernel(float2 *h0, float2 *ht,
		unsigned int in_width, unsigned int out_width,
		unsigned int out_height, float t, float patchSize)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int in_index = y*in_width+x;
    unsigned int in_mindex = (out_height - y)*in_width + (out_width - x); // mirrored
    unsigned int out_index = y*out_width+x;

    float2 k;
    k.x = (-(int)out_width / 2.0f + x) * (2.0f * HIP_PI_F / patchSize);
    k.y = (-(int)out_width / 2.0f + y) * (2.0f * HIP_PI_F / patchSize);

    float k_len = sqrtf(k.x*k.x + k.y*k.y);
    float w = sqrtf(9.81f * k_len);

    if ((x < out_width) && (y < out_height))
    {
        float2 h0_k = h0[in_index];
        float2 h0_mk = h0[in_mindex];
        ht[out_index] = complex_add(complex_mult(h0_k, complex_exp(w * t)),
        		complex_mult(conjugate(h0_mk), complex_exp(-w * t)));
    }
}

// update height map values based on output of FFT
__global__ void updateHeightmapKernel(float  *heightMap,
		float2 *ht, unsigned int width)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int i = y * width  +x;

    float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;

    heightMap[i] = ht[i].x * sign_correction;
}

// generate slope by partial differences in spatial domain
__global__ void calculateSlopeKernel(float2 *slopeOut, float *h, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int i = y * width + x;

    float2 slope = make_float2(0.0f, 0.0f);

    if ((x > 0) && (y > 0) && (x < width-1) && (y < height-1)) {
        slope.x = h[i-1] - h[i+1];
        slope.y = h[i-width] - h[i+width];
    } else {
    	slope.x = slope.y = 0.0;
    }

    slopeOut[i] = slope;
}

// wrapper functions
extern "C" {

void cudaGenerateSpectrumKernel(float2 *d_h0, float2 *d_ht, unsigned int in_width, unsigned int out_width, unsigned int out_height, float animTime, float patchSize)
{
    dim3 block(16, 16, 1);
    dim3 grid((out_width - 1) / block.x + 1, (out_height - 1) / block.y + 1, 1);
    generateSpectrumKernel<<<grid, block>>>(d_h0, d_ht, in_width, out_width, out_height, animTime, patchSize);
}

void cudaUpdateHeightmapKernel(float  *d_heightMap, float2 *d_ht, unsigned int width, unsigned int height)
{
    dim3 block(16, 16, 1);
    dim3 grid((width - 1) / block.x + 1, (height - 1) / block.y + 1, 1);
    updateHeightmapKernel<<<grid, block>>>(d_heightMap, d_ht, width);
}

void cudaCalculateSlopeKernel(float2 *slopeOut, float *hptr, unsigned int width, unsigned int height)
{
    dim3 block(16, 16, 1);
    dim3 grid((width - 1) / block.x + 1, (height - 1) / block.y + 1, 1);
    calculateSlopeKernel<<<grid, block>>>(slopeOut, hptr, width, height);
}

}

